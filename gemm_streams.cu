#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// 행렬 크기 정의
#define N 4096

// CUDA 오류 검사 함수
#define CUDA_CHECK(err) if (err != hipSuccess) { \
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
    exit(EXIT_FAILURE); \
}

// cuBLAS 오류 검사 함수
#define CUBLAS_CHECK(err) if (err != HIPBLAS_STATUS_SUCCESS) { \
    std::cerr << "cuBLAS error: " << err << std::endl; \
    exit(EXIT_FAILURE); \
}

// GEMM 연산 함수
void gemm(hipblasHandle_t handle, hipStream_t stream, float* A, float* B, float* C, int n) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 행렬 곱셈: C = alpha * A * B + beta * C
    hipblasSetStream(handle, stream);
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                             &alpha, A, n, B, n, &beta, C, n));
}

int main() {
    // cuBLAS 핸들 및 CUDA 스트림 생성
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    hipStream_t stream1, stream2;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));

    // 타이머용 CUDA 이벤트 생성
    hipEvent_t start1, stop1, start2, stop2; 
    hipEvent_t start, stop;  // 전체 실행 시간 측정용
    CUDA_CHECK(hipEventCreate(&start1));
    CUDA_CHECK(hipEventCreate(&stop1));
    CUDA_CHECK(hipEventCreate(&start2));
    CUDA_CHECK(hipEventCreate(&stop2));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // 호스트와 디바이스 메모리 할당
    float *h_A, *h_B, *h_C1, *h_C2;
    float *d_A1, *d_B1, *d_C1;
    float *d_A2, *d_B2, *d_C2;
    size_t size = N * N * sizeof(float);

    h_A = (float*)malloc(size);  // 동일한 A 행렬을 사용
    h_B = (float*)malloc(size);  // 동일한 B 행렬을 사용
    h_C1 = (float*)malloc(size); // C1 결과 저장
    h_C2 = (float*)malloc(size); // C2 결과 저장

    CUDA_CHECK(hipMalloc(&d_A1, size));
    CUDA_CHECK(hipMalloc(&d_B1, size));
    CUDA_CHECK(hipMalloc(&d_C1, size));

    CUDA_CHECK(hipMalloc(&d_A2, size));
    CUDA_CHECK(hipMalloc(&d_B2, size));
    CUDA_CHECK(hipMalloc(&d_C2, size));

    // 임의의 행렬 데이터 초기화
    for (int i = 0; i < N * N; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 행렬 데이터를 GPU로 복사 (비동기 복사)
    CUDA_CHECK(hipMemcpyAsync(d_A1, h_A, size, hipMemcpyHostToDevice, stream1));
    CUDA_CHECK(hipMemcpyAsync(d_B1, h_B, size, hipMemcpyHostToDevice, stream1));
    CUDA_CHECK(hipMemcpyAsync(d_A2, h_A, size, hipMemcpyHostToDevice, stream2));
    CUDA_CHECK(hipMemcpyAsync(d_B2, h_B, size, hipMemcpyHostToDevice, stream2));

    // 전체 타이머 시작
    CUDA_CHECK(hipEventRecord(start, 0));

    // 타이머 시작 (stream1)
    auto startTime1 = std::chrono::high_resolution_clock::now(); // 시스템 시간 기록
    CUDA_CHECK(hipEventRecord(start1, stream1));
    gemm(handle, stream1, d_A1, d_B1, d_C1, N);  // 첫 번째 GEMM 연산
    CUDA_CHECK(hipEventRecord(stop1, stream1));

    // 타이머 시작 (stream2)
    auto startTime2 = std::chrono::high_resolution_clock::now(); // 시스템 시간 기록
    CUDA_CHECK(hipEventRecord(start2, stream2));
    gemm(handle, stream2, d_A2, d_B2, d_C2, N);  // 두 번째 GEMM 연산
    CUDA_CHECK(hipEventRecord(stop2, stream2));

    // 두 스트림 동기화
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));

    // 전체 타이머 종료
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));

    // 전체 실행 시간 측정
    float totalTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&totalTime, start, stop));

    // 두 스트림의 실행 시간 측정
    float time1 = 0.0f, time2 = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time1, start1, stop1));
    CUDA_CHECK(hipEventElapsedTime(&time2, start2, stop2));

    // 시작 시점 출력
    auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(startTime1.time_since_epoch()).count();
    auto duration2 = std::chrono::duration_cast<std::chrono::milliseconds>(startTime2.time_since_epoch()).count();
    std::cout << "Stream 1 Start Time: " << duration1 << " ms" << std::endl;
    std::cout << "Stream 2 Start Time: " << duration2 << " ms" << std::endl;

    // 실행 시간 출력
    std::cout << "Total Execution Time: " << totalTime << " ms" << std::endl;
    std::cout << "Stream 1 (GEMM 1) Execution Time: " << time1 << " ms" << std::endl;
    std::cout << "Stream 2 (GEMM 2) Execution Time: " << time2 << " ms" << std::endl;

    // 결과를 GPU에서 호스트로 복사 (비동기 복사)
    CUDA_CHECK(hipMemcpyAsync(h_C1, d_C1, size, hipMemcpyDeviceToHost, stream1));
    CUDA_CHECK(hipMemcpyAsync(h_C2, d_C2, size, hipMemcpyDeviceToHost, stream2));

    // 스트림을 동기화하여 결과 복사 완료를 보장
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));

    // 결과 출력 (일부 값만 출력)
    std::cout << "GEMM 1 Result (C1[0]): " << h_C1[0] << std::endl;
    std::cout << "GEMM 2 Result (C2[0]): " << h_C2[0] << std::endl;

    // 자원 해제
    free(h_A); free(h_B); free(h_C1); free(h_C2);
    CUDA_CHECK(hipFree(d_A1)); CUDA_CHECK(hipFree(d_B1)); CUDA_CHECK(hipFree(d_C1));
    CUDA_CHECK(hipFree(d_A2)); CUDA_CHECK(hipFree(d_B2)); CUDA_CHECK(hipFree(d_C2));

    // 이벤트 및 스트림 삭제
    CUDA_CHECK(hipEventDestroy(start1));
    CUDA_CHECK(hipEventDestroy(stop1));
    CUDA_CHECK(hipEventDestroy(start2));
    CUDA_CHECK(hipEventDestroy(stop2));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));

    return 0;
}