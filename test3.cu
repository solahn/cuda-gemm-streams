#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>

// CUDA 에러 핸들링 매크로
#define CUDA_CHECK(err) if(err != hipSuccess) { \
    std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
    exit(EXIT_FAILURE); \
}

#define CUBLAS_CHECK(err) if(err != HIPBLAS_STATUS_SUCCESS) { \
    std::cerr << "cuBLAS Error: " << err << std::endl; \
    exit(EXIT_FAILURE); \
}

const int N = 4096; // 행렬 크기 (큰 작업)
const int BLOCK_SIZE = 2048; // 작은 작업으로 나눌 크기

// GEMM 수행 함수
void gemm(hipblasHandle_t handle, float* A, float* B, float* C, int n, hipStream_t stream) {
    const float alpha = 1.0f, beta = 0.0f;
    hipblasSetStream(handle, stream);
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n));
}

int main() {
    // cuBLAS 핸들 생성
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // 두 개의 스트림 생성
    hipStream_t stream1, stream2;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));

    // 큰 행렬 할당 (N x N)
    float *A1, *B1, *C1, *A2, *B2, *C2;
    CUDA_CHECK(hipMalloc(&A1, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&B1, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&C1, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&A2, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&B2, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&C2, N * N * sizeof(float)));

    // 큰 작업 - 실행 시간 측정
    auto start = std::chrono::high_resolution_clock::now();
    gemm(handle, A1, B1, C1, N, stream1);
    gemm(handle, A2, B2, C2, N, stream2);
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Large tasks execution time: " 
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() 
              << " ms" << std::endl;

    // 작은 작업으로 나누기 (BLOCK_SIZE x BLOCK_SIZE)
    start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < N; i += BLOCK_SIZE) {
        for (int j = 0; j < N; j += BLOCK_SIZE) {
            gemm(handle, A1 + i * N + j, B1 + i * N + j, C1 + i * N + j, BLOCK_SIZE, stream1);
            gemm(handle, A2 + i * N + j, B2 + i * N + j, C2 + i * N + j, BLOCK_SIZE, stream2);
        }
    }
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));
    end = std::chrono::high_resolution_clock::now();
    std::cout << "Small tasks execution time: " 
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() 
              << " ms" << std::endl;

    // 메모리 해제
    CUDA_CHECK(hipFree(A1));
    CUDA_CHECK(hipFree(B1));
    CUDA_CHECK(hipFree(C1));
    CUDA_CHECK(hipFree(A2));
    CUDA_CHECK(hipFree(B2));
    CUDA_CHECK(hipFree(C2));

    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));

    return 0;
}
